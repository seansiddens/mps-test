#include <stdlib.h>
#include <stdint.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <unistd.h>

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


/* Kernel which does work for some fixed duration of time (specified in milliseconds).
   https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#time-function
   "Sampling this counter at the beginning and at the end of a kernel, taking the difference of the two samples, 
   and recording the result per thread provides a measure for each thread of the number of clock cycles taken by the device 
   to completely execute the thread, but not of the number of clock cycles the device actually spent executing thread instructions. 
   The former number is greater than the latter since threads are time sliced."
*/
__global__ void delay_kernel(float *d_out, float *d_in, int n, uint64_t duration, int clock_rate_khz) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        uint64_t start_clock = clock64();
        uint64_t duration_clocks = (uint64_t)(duration * clock_rate_khz);
        float temp = d_in[idx];
        while (clock64() - start_clock < duration_clocks) {
            temp += sinf(temp);
        }
        d_out[idx] = temp;
    }
}


/** Kernel which does some fixed amount of work. */
__global__ void busy_kernel(float *d_out, float *d_in, int n, uint64_t num_iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    if (idx < n) {
        float temp = d_in[idx];
        // Loop that does some work to prevent compiler optimization
        for (int i = 0; i < 100000000; i++) {
            temp += sinf(temp);
        }
        d_out[idx] = temp;
    }
}

int main(int argc, char *argv[]){
    // Duration for which the kernel should run (in milliseconds).
    uint32_t duration = 1000;

    // Kernel launch params.
    int num_workgroups = 1024;
    int workgroup_size = 256;

    // Scratchpad
    int n = 1024;
    float *h_in = (float*)malloc(n * sizeof(float));
    float *h_out = (float*)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++) {
        h_in[i] = (float)i;
    }

    int pid = getpid();

    // Get device info.
    int device_id = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id); 
    std::cout << "[" << pid << "] Device: " << prop.name << std::endl;
    int clock_rate_khz;
    hipDeviceGetAttribute(&clock_rate_khz, hipDeviceAttributeClockRate, device_id);
    std::cout << "[" << pid << "] Clock rate: " << clock_rate_khz << " kHz" << std::endl;

    // Allocate device memory
    float *d_in, *d_out;
    // uint64_t *d_duration;
    // int *d_clock_rate_khz;
    hipMalloc(&d_in, n * sizeof(float));
    hipMalloc(&d_out, n * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_in, h_in, n * sizeof(float), hipMemcpyHostToDevice);
    // cudaMemcpy(d_duration, &duration, sizeof(uint64_t), cudaMemcpyHostToDevice);
    // cudaMemcpy(d_clock_rate_khz, &clock_rate_khz, sizeof(int), cudaMemcpyHostToDevice);

    // Launch the kernel
    std::cout << "[" << pid << "] Launching kernel" << std::endl;
    auto now = std::chrono::high_resolution_clock::now();
    delay_kernel<<<num_workgroups, workgroup_size>>>(d_out, d_in, n, duration, clock_rate_khz);
    hipDeviceSynchronize();
    cudaCheckErrors("kernel fail");
    auto total_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - now);
    std::cout << "[" << pid << "] Total time (host): " << total_time.count() << " ms" << std::endl;
    
    // Copy result back to host
    hipMemcpy(h_out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}